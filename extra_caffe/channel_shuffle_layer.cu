#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/channel_shuffle_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ChannelShuffleForward(const int n, const Dtype* in, Dtype* out,
    const int bottom_dim, const int feature_dim, const int channels) {
  CUDA_KERNEL_LOOP(index, n) {
    const int n = index / bottom_dim;
    const int i = (index - n*bottom_dim) / feature_dim;
    const int j = index - n*bottom_dim - i*feature_dim;
    const int new_index = n*bottom_dim + j*channels + i; 
    out[new_index] = in[index];
  }
}

template <typename Dtype>
void ChannelShuffleLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  if (shuffle_pattern == 0) {
    // NOLINT_NEXT_LINE(whitespace/operators)
    ChannelShuffleForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, bottom_dim_, feature_dim_, channels_);
    CUDA_POST_KERNEL_CHECK;
  } else {
    // TODO(Shuan) : parallelize Forward and Backward as pattern 0
    for (int n = 0; n < num_; n++) {
        for (int g = 0; g < group_; g++) {
            for (int c = 0; c < group_chnl_num_; c+=shf_chnl_num_) {
                int group_index = (c/shf_chnl_num_)%group_;
                int feature_index = (c/shf_chnl_num_)/group_;
                caffe_copy(
                    shf_chnl_num_*feature_dim_,
                    bottom_data+n*bottom_dim_+g*group_dim_+c*feature_dim_,
                    top_data+n*bottom_dim_+group_index*group_dim_+feature_index*feature_dim_*shf_chnl_num_
                );
            }
        }
    }  
  }
  
}

template <typename Dtype>
__global__ void ChannelShuffleBackward(const int n, const Dtype* in, Dtype* out,
    const int bottom_dim, const int feature_dim, const int channels) {
  CUDA_KERNEL_LOOP(index, n) {
    const int n = index / bottom_dim;
    const int j = (index - n*bottom_dim) / channels;
    const int i = index - n*bottom_dim - j*channels;
    const int new_index = n*bottom_dim + j + i*feature_dim; 
    out[new_index] = in[index];
  }
}

template <typename Dtype>
void ChannelShuffleLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    if (shuffle_pattern == 0) {
        ChannelShuffleBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
            count, top_diff, bottom_diff, bottom_dim_, feature_dim_, channels_);
        CUDA_POST_KERNEL_CHECK;
    } else {
       // TODO(Shuan) : parallelize Forward and Backward as pattern 0
       for (int n = 0; n < num_; n++) {
        for (int g = 0; g < group_; g++) {
            for (int c = 0; c < group_chnl_num_; c+=shf_chnl_num_) {
                int group_index = (c/shf_chnl_num_)%group_;
                int feature_index = (c/shf_chnl_num_)/group_;
                caffe_copy(
                    shf_chnl_num_*feature_dim_,
                    top_diff+n*bottom_dim_+group_index*group_dim_+feature_index*feature_dim_*shf_chnl_num_,
                    bottom_diff+n*bottom_dim_+g*group_dim_+c*feature_dim_
                );
            }
        }
      } 
    }
  }
}
INSTANTIATE_LAYER_GPU_FUNCS(ChannelShuffleLayer);
}  // namespace caffe
